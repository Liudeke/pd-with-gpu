#include "hip/hip_runtime.h"
#include <iostream>

#include <pd/tet_strain_constraint.h>

#include <Eigen/SVD>
#include <Eigen/Geometry>

namespace pd
{
	Eigen::VectorXf TetStrainConstraint::local_solve(const Eigen::VectorXf& q) const
	{
		Eigen::VectorXf ret;
		ret.resize(3);


		return ret;
	}

	TetStrainConstraint::TetStrainConstraint(float wc, const Positions& p, Eigen::RowVector4i vertices) : 
		Constraint(wc, vertices.size())
	{
		assert(n_vertices == 4);
		hipMallocManaged(&this->vertices, sizeof(int) * 4);
		for (int i = 0; i < 4; i++)
		{
			this->vertices[i] = vertices[i];
		}

		precompute_D_m_inv(p, wc);
	}

	TetStrainConstraint::TetStrainConstraint(float wc, const Positions& p, Eigen::RowVector4i vertices, Eigen::Vector3f min_strain_xyz, Eigen::Vector3f max_strain_xyz):
		TetStrainConstraint(wc, p, vertices)
	{
		this->min_strain_xyz = min_strain_xyz;
		this->max_strain_xyz = max_strain_xyz;
	}

	void TetStrainConstraint::precompute_D_m_inv(const Positions& positions, float wc)
	{
		Eigen::Matrix3d D_m;
		const Eigen::Vector3d pivot_vertex_pos = positions.row(vertices[0]).transpose();
		for (int i = 0; i < 3; i++)
		{
			const int v = vertices[i + 1];
			const Eigen::Vector3d cur_vertex_pos = positions.row(v).transpose();
			D_m.col(i) = cur_vertex_pos - pivot_vertex_pos;
		}

		// set weight related to rest volume
		const float volume = static_cast<float>(std::abs(D_m.determinant()) / 6);
		wc *= volume;

		constexpr float EPS = 1e-5;
		if (volume < EPS)
		{
			printf("Tetrahedron %d, %d, %d, %d is too small, the tetrahedralization may be wrong!\n", vertices[0], vertices[1], vertices[2], vertices[3]);
			assert(false);
		}

		// precompute D_m_inv
		D_m_inv = D_m.inverse().cast<float>();
	}

	std::vector<Eigen::Triplet<float>> TetStrainConstraint::get_c_AcTAc(int n_vertex_offset) const
	{
		std::vector<Eigen::Triplet<float>> triplets(3 * 4 * 4); // 48

		int local_max_vertex_idx = 0;
		for (int i = 0; i < n_vertices; i++)
		{
			local_max_vertex_idx = std::max(local_max_vertex_idx, vertices[i]);
		}

		Eigen::SparseMatrix<float> A_c(3, local_max_vertex_idx + 1); // preallocate space is necessary
		A_c.setZero();

		for (int i = 0; i < 4; i++)
		{
			const int v = vertices[i];
			for (int j = 0; j < 3; j++)
			{
				if (i == 0)
				{
					A_c.insert(j, v) = -(D_m_inv.coeff(0, j) + D_m_inv.coeff(1, j) + D_m_inv.coeff(2, j));
				}
				else
				{
					A_c.insert(j, v) = D_m_inv.coeff(i - 1, j);
				}
			}
		}

		A_c.makeCompressed();
		// for (int i = 0; i < n_vertices; i++)
		// {
		// 	std::cout << vertices[i] << "\n";
		// }

		Eigen::SparseMatrix<float> AcT_Ac = wc * A_c.transpose() * A_c;
		AcT_Ac.makeCompressed();
		for (int i = 0; i < AcT_Ac.outerSize(); i++)
		{
			for (Eigen::SparseMatrix<float>::InnerIterator itr(AcT_Ac, i); itr; ++itr)
			{
				for (int j = 0; j < 3; j++)
				{
					triplets.emplace_back(3 * n_vertex_offset + 3 * itr.row() + j, 3 * n_vertex_offset + 3 * itr.col() + j, itr.value());
				}
			}
		}

		// std::cout << "AcT_Ac = " << AcT_Ac << "\n";
		// for (int i = 0; i < triplets.size(); i++)
		// {
		// 	printf("%d, %d, %f\n", triplets[i].row(), triplets[i].col(), triplets[i].value());
		// }
		// printf("triplets.size() = %d\n", triplets.size());
		// assert(false);

		return triplets;
	}

	__host__ __device__ void TetStrainConstraint::project_c_AcTAchpc(float* __restrict__ b, const float* __restrict__ q) const
	{
		#ifndef __CUDA_ARCH__
		// for (int i = 0; i < 3 * 8; i += 3)
		// {
		// 	printf("vertex %d pos = %f %f %f\n", i / 3, q[i], q[i + 1], q[i + 2]);
		// }
		#endif			

		Eigen::Vector3f cur_pos[4];
		for (int i = 0; i < 4; i++)
		{
			cur_pos[i] = { q[3 * vertices[i]], q[3 * vertices[i] + 1], q[3 * vertices[i] + 2] };
		#ifndef __CUDA_ARCH__
			// std::cout << "cur_pos[i] = " << cur_pos[i] << "\n";
		#endif			
		}

		Eigen::Matrix3f D_s;
		for (int i = 0; i < 3; i++)
		{
			D_s.col(i) = cur_pos[i + 1] - cur_pos[0];
		}

		Eigen::Matrix3f F = D_s * D_m_inv; // deformation gradient

	#ifndef __CUDA_ARCH__
		// std::cout << "D_s = " << D_s << "\n";
		// std::cout << "D_m_inv = " << D_m_inv << "\n";
		// std::cout << "deformation gradient = " << F << "\n";
	#endif

		Eigen::Matrix3f Achpc;
	#ifdef __CUDA_ARCH__

	#else
		const bool tet_inverted = F.determinant() < 0;
		Eigen::JacobiSVD<Eigen::Matrix3f> svd(F, Eigen::ComputeFullU | Eigen::ComputeFullV);
		Eigen::Matrix3f U = svd.matrixU();
		Eigen::Matrix3f V = svd.matrixV();
		Eigen::Vector3f sigma = svd.singularValues();
		for (int i = 0; i < 3; i++)
		{
			sigma(i) = std::clamp(sigma(i), min_strain_xyz(i), max_strain_xyz(i));
		}

		// if (tet_inverted)
		// {
		// 	sigma(2) = -sigma(2);
		// }

		Achpc = U * sigma.asDiagonal() * V.transpose();
		// std::cout << "Achpc = " << Achpc << "\n";
	#endif

		// apply A_c^T
		for (int i = 0; i < 4; i++)
		{
			const int v = vertices[i];
			
			Eigen::RowVector3f sum_of_products;
			sum_of_products.setZero();
			for (int j = 0; j < 3; j++)
			{
				// pivot vertex
				if (i == 0)
				{
					Eigen::RowVector3f neg_D_m_inv = -(D_m_inv.row(0) + D_m_inv.row(1) + D_m_inv.row(2));
					sum_of_products += Achpc.row(j) * neg_D_m_inv(j);
				}
				else
				{
					sum_of_products += Achpc.row(j) * D_m_inv.coeff(i - 1, j);
				}
			}

			for (int j = 0; j < 3; j++)
			{
			#ifdef __CUDA_ARCH__
				atomicAdd(&b[3 * v + j], sum_of_products[j] * wc);
			#else			
				b[3 * v + j] += sum_of_products[j] * wc;
				// std::cout << "write val = " << sum_of_products[j] << ", " << wc << "\n";
			#endif
			}
		}
	}
}