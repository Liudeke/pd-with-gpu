#include "hip/hip_runtime.h"
#include <iostream>
#include <pd/bending_constraint.h>

namespace pd
{
    __host__ __device__ BendingConstraint::BendingConstraint(
		float wc, 
		int n_vertices, 
		float rest_mean_curvature,
		int* const vertices,
		float* const laplacian_weights
	): Constraint(wc, n_vertices), rest_mean_curvature(rest_mean_curvature)
	{
		this->vertices = vertices;
		this->laplacian_weights = laplacian_weights;
	}

	BendingConstraint::BendingConstraint(float wc, int center_vertex, const std::vector<int>& neighbor_vertices, const Positions& positions) : Constraint(wc, 1 + neighbor_vertices.size())
	{
		this->vertices = new int[n_vertices];
		vertices[0] = center_vertex;
		for (int i = 0; i < n_vertices - 1; i++)
		{
			vertices[i + 1] = neighbor_vertices[i];
		}

		precompute_laplacian_weights(neighbor_vertices, positions);
	}

	Eigen::VectorXf BendingConstraint::local_solve(const Eigen::VectorXf& q) const
	{
		Eigen::VectorXf ret;
		ret.resize(3);

		// for unit test only


		return ret;
	}

	std::vector<Eigen::Triplet<float>> BendingConstraint::get_c_AcTAc(int n_vertex_offset) const
	{
		std::vector<Eigen::Triplet<float>> triplets(3 * n_vertices - 1);

		const int center_vertex = vertices[0];

		for (int i = 0; i < n_vertices; i++)
		{
			const int v = vertices[i];

			// discard terms between adjacent vertices since they are rather too small
			for (int j = 0; j < 3; j++)
			{
				float val = laplacian_weights[0] * laplacian_weights[i] * wc;
				// printf("%f, %f, %f\n", laplacian_weights[0], laplacian_weights[i], wc);
				// printf("%d-%d adds %f\n", center_vertex, v, val);
				triplets.emplace_back(
					3 * n_vertex_offset + 3 * center_vertex + j, 
					3 * n_vertex_offset + 3 * v + j, 
					val
				);
				if (v != center_vertex)
				{
					triplets.emplace_back(
						3 * n_vertex_offset + 3 * v + j, 
						3 * n_vertex_offset + 3 * center_vertex + j, 
						val
					);
				}
			}
		}

		return triplets;
	}

	__host__ void BendingConstraint::precompute_laplacian_weights(const std::vector<int>& neighbor_vertices, const Positions& positions)
	{
		const int neighbor_size = neighbor_vertices.size();
		assert(neighbor_size >= 3); // avoid singular vertex

		laplacian_weights = new float[n_vertices];
		laplacian_weights[0] = 0.0f; // init value

		const int center_vertex = vertices[0];

		const Eigen::Vector3d center_pos = positions.row(center_vertex).transpose();
		// traverse in counter-clockwise order
		for (int i = 0; i < neighbor_size; i++)
		{
			const int cur_pos_idx = neighbor_vertices[i];
			const int counter_clockwise_next_pos_idx = neighbor_vertices[(i + 1) % neighbor_size];
			const int clockwise_next_pos_idx = neighbor_vertices[(i + neighbor_size - 1) % neighbor_size];
			const Eigen::Vector3d cur_pos = positions.row(cur_pos_idx).transpose();
			const Eigen::Vector3d counter_clockwise_next_pos = positions.row(counter_clockwise_next_pos_idx).transpose();
			const Eigen::Vector3d clockwise_next_pos = positions.row(clockwise_next_pos_idx).transpose();

			if (is_collinear(cur_pos, center_pos, counter_clockwise_next_pos))
			{
				printf("Warning: Vertex %d, %d, %d are collinear, the triangulation of the mesh may be wrong!\n", cur_pos_idx, counter_clockwise_next_pos_idx, center_vertex);
				assert(false);
			}
			if (is_collinear(cur_pos, center_pos, clockwise_next_pos))
			{
				printf("Warning: Vertex %d, %d, %d are collinear, the triangulation of the mesh may be wrong!\n", cur_pos, clockwise_next_pos_idx, center_vertex);
				assert(false);
			}

			const double dis = (cur_pos - center_pos).norm();

			const double tan_alpha = get_half_tan(cur_pos, center_pos, counter_clockwise_next_pos);
			const double tan_beta = get_half_tan(cur_pos, center_pos, clockwise_next_pos);

			const double coefficient = (tan_alpha + tan_beta) / dis;

			laplacian_weights[i + 1] = static_cast<float>(-coefficient);
			laplacian_weights[0] += static_cast<float>(coefficient);
		}

		// Debug only
		// for (int i = 0; i < neighbor_size; i++)
		// {
		// 	printf("%f\n", laplacian_weights[i]);
		// }

		Eigen::Vector3f rest_mean_curvature_vector = apply_laplacian(positions).cast<float>();
		// std::cout << rest_mean_curvature_vector << "\n";
		rest_mean_curvature = rest_mean_curvature_vector.norm();
	}

	__host__ Eigen::Vector3d BendingConstraint::apply_laplacian(const Positions& positions) const
	{
		const int center_vertex = vertices[0];
		const Eigen::Vector3d center_pos = positions.row(center_vertex).transpose();

		// Version 1
		Eigen::Vector3d ret;
		ret.setZero();
		for (int i = 1; i < n_vertices; i++)
		{
			const int cur_pos_idx = vertices[i];
			const Eigen::Vector3d cur_pos = positions.row(cur_pos_idx).transpose();

			ret += (center_pos - cur_pos) * laplacian_weights[i];
		}

		// Version2 (faster)
		Eigen::Vector3d rret;
		rret.setZero();
		for (int i = 0; i < n_vertices; i++)
		{
			const int cur_pos_idx = vertices[i];
			const Eigen::Vector3d cur_pos = positions.row(cur_pos_idx).transpose();
			rret += cur_pos * laplacian_weights[i];
		}

		// std::cout << ret << "\n" << rret << "\n";
		// while (1);

		return ret;
	}

	__host__ __device__ Eigen::Vector3f BendingConstraint::apply_laplacian(const float* __restrict__ q) const
	{
		Eigen::Vector3f ret;
		ret.setZero();

		const int center_vertex = vertices[0];
		const Eigen::Vector3f center_pos = { q[3 * center_vertex], q[3 * center_vertex + 1], q[3 * center_vertex + 2] };

		// TODO: use version 2 to speed up
		for (int i = 1; i < n_vertices; i++)
		{
			const int cur_pos_idx = vertices[i];
			const Eigen::Vector3f cur_pos = { q[3 * cur_pos_idx], q[3 * cur_pos_idx + 1], q[3 * cur_pos_idx + 2] };

			ret += (center_pos - cur_pos) * laplacian_weights[i];
		}
		return ret;
	}

	__host__ __device__ void BendingConstraint::project_c_AcTAchpc(float* __restrict__ b, const float* __restrict__ q) const
	{
		const float EPS = 1e-5;
	
		if (std::abs(rest_mean_curvature) < EPS)
		{
			return; // no constraint indeed
		}
		printf("Rest mean curvature = %f\n", rest_mean_curvature);

		Eigen::Vector3f deformed_laplace = apply_laplacian(q);
		const float deformed_laplace_norm = deformed_laplace.norm();

		Eigen::Vector3f Achpc;
		if (deformed_laplace_norm < EPS)
		// if (true)
		{
			// if norm too small, don't divide by it, instead just use current normal
			Achpc = get_center_vertex_normal(q) * rest_mean_curvature; // mean curvature vector
		}
		else 
		{
			Achpc = deformed_laplace * rest_mean_curvature / deformed_laplace_norm; // Ac'pc = v_f * |v_g| / |v_f|
		}

		for (int i = 0; i < n_vertices; i++)
		{
			const int v = vertices[i];
			for (int j = 0; j < 3; j++)
			{
			#ifdef __CUDA_ARCH__
				atomicAdd(&b[3 * v + j], wc * laplacian_weights[i] * Achpc[j]);
			#else			
				b[3 * v + j] += wc * laplacian_weights[i] * Achpc[j];
			#endif
			}
		}
	}

	__host__ __device__ Eigen::Vector3f BendingConstraint::get_center_vertex_normal(const float* __restrict__ q) const
	{
		const int neighbor_size = n_vertices - 1;

		const int center_vertex = vertices[0];

		const Eigen::Vector3f center_pos = { q[3 * center_vertex], q[3 * center_vertex + 1], q[3 * center_vertex + 2] };

		Eigen::Vector3f normal;
		normal.setZero();
		for (int i = 0; i < neighbor_size; i++)
		{
			const int cur_pos_idx = vertices[i + 1];
			const int counter_clockwise_next_pos_idx = vertices[((i + 1) % neighbor_size) + 1];
			const Eigen::Vector3f cur_pos = { q[3 * cur_pos_idx], q[3 * cur_pos_idx + 1], q[3 * cur_pos_idx + 2] };
			const Eigen::Vector3f counter_clockwise_next_pos = { q[3 * counter_clockwise_next_pos_idx], q[3 * counter_clockwise_next_pos_idx + 1], q[3 * counter_clockwise_next_pos_idx + 2] };

			normal += get_triangle_normal(counter_clockwise_next_pos - center_pos, cur_pos - center_pos);
		}
		normal.normalize(); // take the average


		// TODO: checkout why this outputs nan 
		// #ifndef __CUDA_ARCH__
		// std::cout << normal << "\n";
		// #endif

		return normal;
	}

	__host__ __device__ Eigen::Vector3f BendingConstraint::get_triangle_normal(Eigen::Vector3f p21, Eigen::Vector3f p31)
	{
		return p21.cross(p31).normalized();
	}
}