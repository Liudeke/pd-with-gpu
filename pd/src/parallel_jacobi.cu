#include "hip/hip_runtime.h"
#include <pd/parallel_jacobi.h>
#include <util/cpu_timer.h>
#include <util/hip/hip_runtime_api.h>
#include <iostream>

namespace pd
{
	__global__ void itr_shfl_down(float* __restrict__ next_x, const float* __restrict__ A, const float* __restrict__ x, const float* __restrict__ b, int n_row, int n_col)
	{
		int col_start = threadIdx.x; // indicates i-th thread in a warp, 0 <= i <= 31
		int row = blockIdx.x;
		int offset = row * n_col;
		float sum = 0.0f;

		if (row < n_row)
		{
			for (int i = col_start; i < n_col; i += blockDim.x) // blockDim.x == 32 == WARP_SIZE
			{
				sum += A[offset + i] * x[i];
			}

			sum += __shfl_down_sync(__activemask(), sum, 16);
			sum += __shfl_down_sync(__activemask(), sum, 8);
			sum += __shfl_down_sync(__activemask(), sum, 4);
			sum += __shfl_down_sync(__activemask(), sum, 2);
			sum += __shfl_down_sync(__activemask(), sum, 1);
			if (threadIdx.x == 0)
			{
				// let the first thread of a warp write next_x
				next_x[row] = (b[row] - (sum - A[offset + row] * x[row])) / A[offset + row];
			}
		}
	}

	__global__ void itr_normal(float* __restrict__ next_x, const float* __restrict__ A, const float* __restrict__ x, const float* __restrict__ b, int n_row, int n_col)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;

		if (idx < n_row)
		{
			float sum = 0.0f;
			int row_offset = idx * n_col;
			for (int j = 0; j < n_col; j++)
			{
				sum += A[row_offset + j] * x[j];
			}
			sum -= A[row_offset + idx] * x[idx];
			next_x[idx] = (b[idx] - sum) / A[row_offset + idx];
		}
	}

	void ParallelJacobi::clear()
	{
		if (is_allocated)
		{
			checkCudaErrors(hipFree(d_A));
			checkCudaErrors(hipFree(d_b));
			checkCudaErrors(hipFree(d_x));
			checkCudaErrors(hipFree(d_next_x));
			is_allocated = false;
		}
	}

	// Make sure A is compressed
	void ParallelJacobi::set_A(const Eigen::SparseMatrix<float>& A, const pd::Constraints& constraints)
	{
		Eigen::MatrixXf _A = Eigen::MatrixXf(A);
		n = _A.rows(); // n = 3 * #Vertex
		//std::cout << "n = " << n << "\n";
		//for (int i = 0; i < _A.rows(); i++)
		//{
		//	std::cout << _A.coeff(i, i) << "\n";
		//}
		checkCudaErrors(hipMalloc((void**)&d_A, sizeof(float) * _A.size()));
		checkCudaErrors(hipMalloc((void**)&d_b, sizeof(float) * n));
		checkCudaErrors(hipMalloc((void**)&d_x, sizeof(float) * n));
		checkCudaErrors(hipMalloc((void**)&d_next_x, sizeof(float) * n));
		is_allocated = true;

		checkCudaErrors(hipMemcpy(d_A, _A.data(), sizeof(float) * _A.size(), hipMemcpyHostToDevice));
	}

	Eigen::VectorXf ParallelJacobi::solve(const Eigen::VectorXf& b)
	{
		Eigen::VectorXf ret;
		ret.resizeLike(b);

		assert(b.size() == n);

		checkCudaErrors(hipMemcpy(d_b, b.data(), sizeof(float) * n, hipMemcpyHostToDevice));
		// set to IEEE-754 zero as iteration initial value
		hipMemset(d_x, 0, sizeof(float) * n);
		hipMemset(d_next_x, 0, sizeof(float) * n);

		// The solver iterates for a constant number, not checking error
		//float diff = 0.0f;
		//float eps = 1e-4f;
		if (false)
		{
			const int n_blocks = n / WARP_SIZE + (n % WARP_SIZE == 0 ? 0 : 1);
			for (int i = 0; i < n_itr; i++)
			{
				// double buffer
				if (i % 2 == 1)
				{
					itr_normal << <n_blocks, WARP_SIZE >> > (d_x, d_A, d_next_x, d_b, n, n);
				}
				else
				{
					itr_normal << <n_blocks, WARP_SIZE >> > (d_next_x, d_A, d_x, d_b, n, n);
				}
				//hipDeviceSynchronize(); no need to call since kernel execution in GPU is sequential
			}
		}
		else
		{
			for (int i = 0; i < n_itr; i++)
			{
				// double buffer
				if (i % 2 == 1)
				{
					itr_shfl_down << <n, WARP_SIZE >> > (d_x, d_A, d_next_x, d_b, n, n);
				}
				else
				{
					itr_shfl_down << <n, WARP_SIZE >> > (d_next_x, d_A, d_x, d_b, n, n);
				}
			}
		}
		//hipDeviceSynchronize();  // no need to call since hipMemcpy is synchronized

		checkCudaErrors(hipMemcpy(ret.data(), d_x, sizeof(float) * n, hipMemcpyDeviceToHost));

		// check if the error is OK
		Eigen::VectorXf err_checker;
		err_checker.resizeLike(ret);
		checkCudaErrors(hipMemcpy(err_checker.data(), d_next_x, sizeof(float) * n, hipMemcpyDeviceToHost));
		constexpr float eps = 1e-3f;
		for (int i = 0; i < n; i++)
		{
			if (std::abs(err_checker[i] - ret[i]) > eps)
			{
				printf("Warning: Jacobi Iteration Incomplete. At index %d, values are %f, %f.\n", i, err_checker[i], ret[i]);
				break;
			}
		}
		// if (true)
		// 	std::cout << "err checker[32] = " << err_checker[32] << "\n" << "ret[32] = " << ret[32] << "\n";

		return ret;
	}
}