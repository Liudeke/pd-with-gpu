#include "hip/hip_runtime.h"
#include <array>
#include <pd/positional_constraint.h>

namespace pd {
	__host__ __device__ PositionalConstraint::PositionalConstraint(float wc, int vi, float x0, float y0, float z0) :
		Constraint(wc, 1, new int[1] {vi}),
		vi(vi),
		x0(x0), y0(y0), z0(z0)
	{
	}

	Eigen::VectorXf PositionalConstraint::local_solve(const Eigen::VectorXf& q) const
	{
		return Eigen::Vector3f(x0, y0, z0);
	}

	std::vector<Eigen::Triplet<float>> PositionalConstraint::get_c_AcTAc(int n_vertex_offset) const
	{
		std::array<Eigen::Triplet<float>, 3u> triplets;
		for (int i = 0; i < 3; i++)
		{
			triplets[i] = { 3 * n_vertex_offset + 3 * vi + i, 3 * n_vertex_offset + 3 * vi + i, wc };
		}
		return std::vector<Eigen::Triplet<float>>{ triplets.begin(), triplets.end() };
	}

	__host__ __device__ void PositionalConstraint::project_c_AcTAchpc(float* __restrict__ b, const float* __restrict__ q) const
	{
#ifdef __CUDA_ARCH__
		atomicAdd(&b[3 * vi], wc * x0);
		atomicAdd(&b[3 * vi + 1], wc * y0);
		atomicAdd(&b[3 * vi + 2], wc * z0);
#else
		b[3 * vi] += wc * x0;
		b[3 * vi + 1] += wc * y0;
		b[3 * vi + 2] += wc * z0;
#endif
	}
}