#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <Eigen/SparseCore>
#include <vector>
#include <array>
#include <iostream>

#include <pd/edge_strain_constraint.h>

namespace pd {
	__host__ __device__ EdgeStrainConstraint::EdgeStrainConstraint(float wc, int vi, int vj, float rest_length) :
		Constraint(wc, 2, new int[2] {vi, vj}),
		vi(vi),
		vj(vj),
		rest_length(rest_length)
	{
		assert(vi != vj);
	}

	Eigen::VectorXf EdgeStrainConstraint::local_solve(const Eigen::VectorXf& q) const
	{
		Eigen::VectorXf ret;
		ret.resize(3);

		// printf("%d %d\n", vi, vj);

		Eigen::Vector3f vi_pos = q.block(3 * vi, 0, 3, 1);
		Eigen::Vector3f vj_pos = q.block(3 * vj, 0, 3, 1);
		Eigen::Vector3f j2i = vj_pos - vi_pos;

		float delta_x = j2i.norm() - rest_length; // This is the constraint set C_i (edge length)
		Eigen::Vector3f j2i_normalized = j2i.normalized();
		
		Eigen::Vector3f pc1 = vi_pos + 0.5f * delta_x * j2i_normalized;
		Eigen::Vector3f pc2 = vj_pos - 0.5f * delta_x * j2i_normalized;

		ret = pc1 - pc2;

		//std::cout << "ret = " << ret << "\n";
		return ret;
	}

	std::vector<Eigen::Triplet<float>> EdgeStrainConstraint::get_c_AcTAc(int n_vertex_offset) const
	{
		std::vector<Eigen::Triplet<float>> triplets(12);

		for (int i = 0; i < 3; i++)
		{
			triplets.emplace_back(3 * n_vertex_offset + 3 * vi + i, 3 * n_vertex_offset + 3 * vi + i, wc);
		}
		for (int i = 0; i < 3; i++)
		{
			triplets.emplace_back(3 * n_vertex_offset + 3 * vj + i, 3 * n_vertex_offset + 3 * vj + i, wc);
		}
		for (int i = 0; i < 3; i++)
		{
			triplets.emplace_back(3 * n_vertex_offset + 3 * vi + i, 3 * n_vertex_offset + 3 * vj + i, -wc);
		}
		for (int i = 0; i < 3; i++)
		{
			triplets.emplace_back(3 * n_vertex_offset + 3 * vj + i, 3 * n_vertex_offset + 3 * vi + i, -wc);
		}

		return std::vector<Eigen::Triplet<float>>{ triplets.begin(), triplets.end() };
	}

	__host__ __device__ void EdgeStrainConstraint::project_c_AcTAchpc(float* __restrict__ b, const float* __restrict__ q) const
	{
		// #vertex offset is already included
		Eigen::Vector3f vi_pos{ q[3 * vi], q[3 * vi + 1], q[3 * vi + 2] };
		Eigen::Vector3f vj_pos{ q[3 * vj], q[3 * vj + 1], q[3 * vj + 2] };

		Eigen::Vector3f j2i = vi_pos - vj_pos;
		Eigen::Vector3f Achpc = j2i / j2i.norm() * rest_length;
		// float delta_x = j2i.norm() - rest_length; // This is the constraint set C_i (edge length)
		// Eigen::Vector3f j2i_normalized = j2i.normalized();

		// Eigen::Vector3f pc1 = vi_pos + 0.5f * delta_x * j2i_normalized;
		// Eigen::Vector3f pc2 = vj_pos - 0.5f * delta_x * j2i_normalized;
		// Eigen::Vector3f Acpc = pc1 - pc2;

		for (int i = 0; i < 3; i++)
		{
		#ifdef __CUDA_ARCH__
			atomicAdd(&b[3 * vi + i], wc * Achpc[i]);
			atomicAdd(&b[3 * vj + i], -wc * Achpc[i]);
		#else
			b[3 * vi + i] += wc * Achpc[i];
			b[3 * vj + i] += -wc * Achpc[i];
		#endif
		}
	}
}